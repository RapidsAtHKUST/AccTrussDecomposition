#include "hip/hip_runtime.h"
#include <climits>
#include <cassert>

#include <chrono>
#include <sstream>
#include <fstream>

#include <omp.h>

#include "util/graph.h"
#include "util/log.h"
#include "util/pretty_print.h"
#include "util/timer.h"
#include "util/cuda/cuda_util.h"

#include "reordering_utils/reorder_utils.h"

#include "cuda_extern_functions.cuh"
#include "main_helper.h"

int main(int argc, char *argv[]) {
    setlocale(LC_NUMERIC, "");
    if (argc < 2) {
        fprintf(stderr, "%s <Graph file>\n", argv[0]);
        exit(1);
    }

    ZLCUDAMemStat mem_stat;
    ZLCUDATimer time_stat;

    string algorithm_name;
    string cmd(argv[0]);
    auto pos = cmd.find_last_of('/');
    if (pos != string::npos) {
        algorithm_name = cmd.substr(pos + 1, cmd.size());
    } else {
        algorithm_name = cmd;
    }
    //set log file descriptor
#ifdef USE_LOG
    FILE *log_f = nullptr;
    if (argc >= 4) {
        log_f = fopen(argv[3], "a+");
        log_set_fp(log_f);
    }
#endif
    log_info("Algorithm name: %s", algorithm_name.c_str());
    read_env();

    graph_t g;

    //load the graph from file
    Graph yche_graph(argv[1]);
    g.adj = yche_graph.edge_dst;
    g.num_edges = yche_graph.node_off;
    g.n = yche_graph.nodemax;
    g.m = yche_graph.edgemax;

    string reorder_method(argv[2]);

    vector<int32_t> new_vid_dict;
    vector<int32_t> old_vid_dict;
    ReorderWrapper(g, argv[1], reorder_method, new_vid_dict, old_vid_dict);

    /************   Compute k - truss *****************************************/
    //edge list array
    Timer get_eid_timer;
    auto edgeIdToEdge = (Edge *) malloc(sizeof(Edge) * (g.m / 2));
    auto EdgeSupport = (int *) malloc(sizeof(int) * (g.m / 2));
    log_info("Malloc Time: %.9lf s", get_eid_timer.elapsed());
    auto org_num_edges = g.m / 2;

    get_eid_timer.reset();
    //Populate the edge list array
    getEidAndEdgeList(&g, edgeIdToEdge);
    log_info("Init Eid Time: %.9lf s", get_eid_timer.elapsed());
    get_eid_timer.reset();

    auto max_omp_threads = omp_get_max_threads();
    log_info("Max Threads: %d", max_omp_threads);
#pragma omp parallel for
    for (auto i = 0; i < max_omp_threads; i++) {
        auto avg = g.m / 2 / max_omp_threads;
        auto iter_beg = avg * i;
        auto iter_end = (i == max_omp_threads - 1) ? g.m / 2 : avg * (i + 1);
        memset(EdgeSupport + iter_beg, 0, (iter_end - iter_beg) * sizeof(int));
    }
    log_info("Init EdgeSupport Time: %.9lf s", get_eid_timer.elapsed());
    get_eid_timer.reset();

    Timer global_timer;
    int *EdgeSupportCUDA, *output, *final_result_output;
    ZLCudaMalloc(&output, sizeof(int) * org_num_edges, &mem_stat);
    ZLCudaMalloc(&final_result_output, sizeof(int) * org_num_edges, &mem_stat);
    ZLCudaMalloc(&EdgeSupportCUDA, sizeof(int) * (g.m / 2), &mem_stat);
    log_info("Malloc Time: %.9lfs", global_timer.elapsed());

#ifndef GPU_ONLY
    eid_t *level_start_pos, *edge_offsets_level, *edge_off_org;
    int *edge_sup;
    Edge *edge_lst;
    auto level = TrussDecompositionLevelsCPU(g, EdgeSupport, edgeIdToEdge,
                                             level_start_pos, edge_offsets_level, edge_off_org,
                                             edge_sup, edge_lst);
    Timer offload_timer;
    hipMemcpy(output, edge_offsets_level, sizeof(int) * org_num_edges, hipMemcpyHostToDevice);
    hipMemcpy(EdgeSupportCUDA, edge_sup, sizeof(int) * g.m / 2, hipMemcpyHostToDevice);

    assert(level_start_pos[level + 1] == 0);
    log_info("Current g.m: %'lld", g.m / 2);
    PKT_cuda(&g, edge_off_org, EdgeSupportCUDA, edge_lst,
             100, output, level_start_pos, &mem_stat, &time_stat, level);
    hipDeviceSynchronize();
    log_info("Offloading Comp Time: %.9lfs", offload_timer.elapsed());
#else
    auto level_start_pos = (eid_t *) calloc(MAX_LEVEL, sizeof(eid_t));
    hipMemcpy(EdgeSupportCUDA, EdgeSupport, sizeof(int) * g.m / 2, hipMemcpyHostToDevice);

    Timer tc_timer;
    invoke_tc_bmp_gpu(&g, EdgeSupportCUDA);
    extern double tc_time;
    tc_time = tc_timer.elapsed();
    PKT_cuda(&g, nullptr, EdgeSupportCUDA, edgeIdToEdge,
             100, output, level_start_pos, &mem_stat, &time_stat, 0);
#endif
    log_info("Parallel K-Truss: %.9lfs", global_timer.elapsed());

    auto output_dir = string(argv[1]) + "/" + string("ktruss-") + algorithm_name + ".histogram";
    log_info("Output Dir: %s", output_dir.c_str());


    /*Recover the EdgeSupport for checking*/
    Timer recovery_timer;
    CheckLevelOff(level_start_pos);
#pragma omp parallel
    for (int l = 0;; l++) {

        auto start = level_start_pos[l];
        auto end = level_start_pos[l + 1];
        if (start == org_num_edges) break;
        if (start > end) {
            log_fatal("error in level stat pos: %d, [%d, %d)", l, start, end);
            exit(-1);
        }
#pragma omp for
        for (int i = start; i < end; i++) {
            final_result_output[output[i]] = l;
        }
    }
    log_info("Finish Constructing Final Results..., Recovery Time: %.9lfs", recovery_timer.elapsed());
    display_stats(final_result_output, org_num_edges, output_dir);
    log_info("Finish Checking, ET: %.9lfs", recovery_timer.elapsed_and_reset());

    //Free memory
    free_graph(&g);
    free(level_start_pos);
#ifndef GPU_ONLY
    free(edge_offsets_level);
    free(edge_off_org);
    free(edge_sup);
    free(edge_lst);
#endif

    free(edgeIdToEdge);
    free(EdgeSupport);
    ZLCudaFree(output, &mem_stat);
    ZLCudaFree(final_result_output, &mem_stat);

    log_info("Free Time: %.9lfs", recovery_timer.elapsed());
#ifdef USE_LOG
    if (log_f != nullptr) {
        log_info("Flush File and Close...");
        fflush(log_f);
        fclose(log_f);
    }
#endif
    return 0;
}
