//
// Created by Bryan on 22/7/2019.
//

#include <cassert>
#include <hip/hip_cooperative_groups.h>
#include <cuda_occupancy.h>

#include "cuda_util.h"

/*memory statistics*/
ZLCUDAMemStat::ZLCUDAMemStat() {
    cur_use = 0;
    max_use = 0;
    acc_use = 0;
    valid = true;
}

void ZLCUDAMemStat::malloc_mem_stat(unsigned long addr, long add_bytes) {
    if (!valid) return;

    cur_use += add_bytes;
    acc_use += add_bytes;
    if (cur_use > max_use) {
        max_use = cur_use;
    }

    /*record the allocated address and memory size*/
    addr_bytes.insert(std::make_pair(addr, add_bytes));
}

void ZLCUDAMemStat::delete_mem_stat(unsigned long addr) {
    if (!valid) return;

    auto iter = addr_bytes.find(addr);
    if (iter == addr_bytes.end()) {
        log_warn("MemStat does not match. The Mem stat becomes invalid.");
        valid = false;
        return;
    }

    long delete_bytes = iter->second;
    addr_bytes.erase(iter); //remove the address
    cur_use -= delete_bytes;
    assert(cur_use >= 0);
}

long ZLCUDAMemStat::get_max_use() {
    return max_use;
}

long ZLCUDAMemStat::get_acc_use() {
    return acc_use;
}

long ZLCUDAMemStat::get_cur_use() {
    return cur_use;
}

void ZLCUDAMemStat::reset() {
    cur_use = 0;
    max_use = 0;
    acc_use = 0;

    addr_bytes.clear();
    valid = true;
}

uint32_t ZLCUDAMemStat::get_num_obj() {
    return (uint32_t)addr_bytes.size();
}

void ZLCUDAMemStat::displaceMap() {
    if (!valid) return;
    log_info("Display the memory map.");
    for(auto i : addr_bytes)
        log_info("%p %ld.", i.first, i.second);
}

/*time statistics*/
ZLCUDATimer::ZLCUDATimer() {
    idx = 0;
}

void ZLCUDATimer::reset() {
    idx = (uint32_t)kernel_time.size();
}

float ZLCUDATimer::elapsed() {
    uint32_t end = (uint32_t)kernel_time.size();
    float res = 0.0;
    for(auto i = idx; i < end; i++)
        res += this->kernel_time[i];
    return res;
}

void ZLCUDATimer::insert_record( std::string file,
                    std::string host_func,
                    std::string kernel,
                    float ker_time) {
    file_name.emplace_back(file);
    host_func_name.emplace_back(host_func);
    kernel_name.emplace_back(kernel);
    kernel_time.emplace_back(ker_time);
}

void check(hipError_t code, char const *const func, const char *const file,
           int const line, bool abort)
{
    if (code != hipSuccess)
    {
        if (abort)
        {
            log_fatal("CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
                      static_cast<unsigned int>(code), hipGetErrorString(code), func);
            // Make sure we call CUDA Device Reset before exiting
            hipDeviceReset();
            exit(static_cast<unsigned int>(code));
        }
        else
        {
            log_warn("CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
                     static_cast<unsigned int>(code), hipGetErrorString(code), func);
        }
    }
}

//void ZLCudaMalloc(void *addr, size_t malloc_bytes, ZLCUDAMemStat *stat)
//{
//    checkCudaErrors(hipMallocManaged((void**)addr, malloc_bytes));
//    checkCudaErrors(hipMemset(*addr, 0, malloc_bytes));
//    if (stat)
//        stat->malloc_mem_stat((unsigned long)*addr, malloc_bytes);
//}
//
//void ZLCudaFree(void *addr, ZLCUDAMemStat *stat)
//{
//    checkCudaErrors(hipFree(addr));
//    if (stat)
//        stat->delete_mem_stat((unsigned long)addr);
//}

