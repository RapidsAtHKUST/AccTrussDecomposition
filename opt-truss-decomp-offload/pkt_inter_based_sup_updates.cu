#include "hip/hip_runtime.h"
#include "util/cuda/cuda_util.h"
#include "cuda_extern_functions.cuh"


__global__ void bmp_update_next(uint32_t *d_offsets, int32_t *d_dsts,
                                uint32_t *d_bitmaps, uint32_t *d_bitmap_states,
                                uint32_t *vertex_count, uint32_t conc_blocks_per_SM,
                                eid_t *eid, int32_t *d_intersection_count_GPU,
                                int val_size_bitmap, int val_size_bitmap_indexes,
                                int level, int *next, int *next_cnt, bool *inNext,
                                InBucketWinType *in_bucket_window_, eid_t *bucket_buf_, uint32_t *window_bucket_buf_size_,
                                int bucket_level_end_
) {
    const uint32_t tid = threadIdx.x + blockDim.x * threadIdx.y; /*threads in a warp are with continuous threadIdx.x */
    const uint32_t tnum = blockDim.x * blockDim.y;
    const uint32_t elem_bits = sizeof(uint32_t) * 8; /*#bits in a bitmap element*/

    __shared__ uint32_t node_id, sm_id, bitmap_ptr;
    __shared__ uint32_t start_src, end_src, start_src_in_bitmap, end_src_in_bitmap;

    extern __shared__ uint32_t bitmap_indexes[];

    if (tid == 0) {
        node_id = atomicAdd(vertex_count, 1); /*get current vertex id*/
        start_src = d_offsets[node_id];
        end_src = d_offsets[node_id + 1];
        start_src_in_bitmap = d_dsts[start_src] / elem_bits;
        end_src_in_bitmap = (start_src == end_src) ? d_dsts[start_src] / elem_bits : d_dsts[end_src - 1] / elem_bits;
    } else if (tid == tnum - 1) {
        uint32_t temp = 0;
        asm("mov.u32 %0, %smid;" : "=r"(sm_id));
        /*get current SM*/
        while (atomicCAS(&d_bitmap_states[sm_id * conc_blocks_per_SM + temp], 0, 1) != 0)
            temp++;
        bitmap_ptr = temp;
    }
    /*initialize the 2-level bitmap*/
    for (uint32_t idx = tid; idx < val_size_bitmap_indexes; idx += tnum)
        bitmap_indexes[idx] = 0;
    __syncthreads();

    uint32_t *bitmap = &d_bitmaps[val_size_bitmap * (conc_blocks_per_SM * sm_id + bitmap_ptr)];

    /*construct the source node neighbor bitmap*/
    for (uint32_t idx = start_src + tid; idx < end_src; idx += tnum) {
        uint32_t src_nei = d_dsts[idx];
        const uint32_t src_nei_val = src_nei / elem_bits;
        atomicOr(&bitmap[src_nei_val], (0b1 << (src_nei & (elem_bits - 1)))); /*setting the bitmap*/
        atomicOr(&bitmap_indexes[src_nei_val >> BITMAP_SCALE_LOG],
                 (0b1 << ((src_nei >> BITMAP_SCALE_LOG) & (elem_bits - 1)))); /*setting the bitmap index*/
    }
    __syncthreads();

    /*loop the neighbors*/
    /* x dimension: warp-size
     * y dimension: number of warps
     * */
    auto du = d_offsets[node_id + 1] - d_offsets[node_id];
    for (uint32_t idx = start_src + threadIdx.y; idx < end_src; idx += blockDim.y) {
        /*each warp processes a node*/
        uint32_t private_count = 0;
        uint32_t src_nei = d_dsts[idx];
        auto dv = d_offsets[src_nei + 1] - d_offsets[src_nei];
        if (dv > du || ((du == dv) && node_id > src_nei))continue;
        uint32_t start_dst = d_offsets[src_nei];
        uint32_t end_dst = d_offsets[src_nei + 1];
        for (uint32_t dst_idx = start_dst + threadIdx.x; dst_idx < end_dst; dst_idx += blockDim.x) {
            uint32_t dst_nei = d_dsts[dst_idx];
            const uint32_t dst_nei_val = dst_nei / elem_bits;
            if ((bitmap_indexes[dst_nei_val >> BITMAP_SCALE_LOG] >> ((dst_nei >> BITMAP_SCALE_LOG) & (elem_bits - 1)))
                & 0b1 == 1)
                if ((bitmap[dst_nei_val] >> (dst_nei & (elem_bits - 1))) & 0b1 == 1)
                    private_count++;
        }
        __syncwarp();
        /*warp-wise reduction*/
        WARP_REDUCE(private_count);
        if (threadIdx.x == 0) {
            auto edge_idx = eid[idx];
            if (private_count < level) {
                private_count = level;
            }
            d_intersection_count_GPU[edge_idx] = private_count;
            if (private_count == level) {
                auto insert_idx = atomicAdd(next_cnt, 1);
                next[insert_idx] = edge_idx;
                inNext[edge_idx] = true;
            }
#ifndef LEGACY_SCAN
            // Update the Bucket.
            auto latest = private_count;
            if (latest > level && latest < bucket_level_end_) {
                auto old_token = atomicCAS(in_bucket_window_ + edge_idx, InBucketFalse, InBucketTrue);
                if (!old_token) {
                    auto insert_idx = atomicAdd(window_bucket_buf_size_, 1);
                    bucket_buf_[insert_idx] = edge_idx;
                }
            }
#endif
        }
    }
    __syncthreads();

    /*clean the bitmap*/
    if (end_src_in_bitmap - start_src_in_bitmap + 1 <= end_src - start_src) {
        for (uint32_t idx = start_src_in_bitmap + tid; idx <= end_src_in_bitmap; idx += tnum) {
            bitmap[idx] = 0;
        }
    } else {
        for (uint32_t idx = start_src + tid; idx < end_src; idx += tnum) {
            uint32_t src_nei = d_dsts[idx];
            bitmap[src_nei / elem_bits] = 0;
        }
    }
    __syncthreads();

    /*release the bitmap lock*/
    if (tid == 0)
        atomicCAS(&d_bitmap_states[sm_id * conc_blocks_per_SM + bitmap_ptr], 1, 0);
}

__global__ void bmp_bsr_update_next(uint32_t *d_offsets, int32_t *d_dsts,
                                    uint32_t *d_bitmaps, uint32_t *d_bitmap_states,
                                    uint32_t *vertex_count, uint32_t conc_blocks_per_SM,
                                    eid_t *eid, int32_t *d_intersection_count_GPU,
                                    int val_size_bitmap, int val_size_bitmap_indexes,
                                    uint32_t *bmp_offs, bmp_word_idx_type *bmp_word_indices, bmp_word_type *bmp_words,
                                    int level, int *next, int *next_cnt, bool *inNext,
                                    InBucketWinType *in_bucket_window_, eid_t *bucket_buf_, uint32_t *window_bucket_buf_size_,
                                    int bucket_level_end_
) {
    const uint32_t tid = threadIdx.x + blockDim.x * threadIdx.y; /*threads in a warp are with continuous threadIdx.x */
    const uint32_t num_threads = blockDim.x * blockDim.y;
    const uint32_t elem_bits = sizeof(uint32_t) * 8; /*#bits in a bitmap element*/

    __shared__ uint32_t u, sm_id, bitmap_ptr;
    __shared__ uint32_t off_u, off_u_plus_one, start_src_in_bitmap, end_src_in_bitmap;

    extern __shared__ uint32_t bitmap_indexes[];

    if (tid == 0) {
        u = atomicAdd(vertex_count, 1); /*get current vertex id*/
        off_u = d_offsets[u];
        off_u_plus_one = d_offsets[u + 1];
        start_src_in_bitmap = d_dsts[off_u] / elem_bits;
        end_src_in_bitmap = (off_u == off_u_plus_one) ? d_dsts[off_u] / elem_bits :
                            d_dsts[off_u_plus_one - 1] / elem_bits;
    } else if (tid == num_threads - 1) {
        uint32_t temp = 0;
        asm("mov.u32 %0, %smid;" : "=r"(sm_id));
        /*get current SM*/
        while (atomicCAS(&d_bitmap_states[sm_id * conc_blocks_per_SM + temp], 0, 1) != 0)
            temp++;
        bitmap_ptr = temp;
    }
    /*initialize the 2-level bitmap*/
    for (uint32_t idx = tid; idx < val_size_bitmap_indexes; idx += num_threads)
        bitmap_indexes[idx] = 0;
    __syncthreads();

    uint32_t *bitmap = &d_bitmaps[val_size_bitmap * (conc_blocks_per_SM * sm_id + bitmap_ptr)];

    /*construct the source node neighbor bitmap*/
    for (uint32_t idx = off_u + tid; idx < off_u_plus_one; idx += num_threads) {
        uint32_t v = d_dsts[idx];
        const uint32_t src_nei_val = v / elem_bits;
        atomicOr(&bitmap[src_nei_val], (0b1 << (v & (elem_bits - 1)))); /*setting the bitmap*/
        atomicOr(&bitmap_indexes[src_nei_val >> BITMAP_SCALE_LOG],
                 (0b1 << ((v >> BITMAP_SCALE_LOG) & (elem_bits - 1)))); /*setting the bitmap index*/
    }
    __syncthreads();

    auto du = d_offsets[u + 1] - d_offsets[u];
    for (uint32_t idx = off_u + threadIdx.y; idx < off_u_plus_one; idx += blockDim.y) {
        uint32_t v = d_dsts[idx];

        /*each warp processes an edge (u, v), v: v */
        auto dv = d_offsets[v + 1] - d_offsets[v];
        if (dv > du || ((du == dv) && u > v))continue;

        uint32_t private_count = 0;
        auto size_nv = bmp_offs[v + 1] - bmp_offs[v];
        if (size_nv > 0) {
            for (uint32_t wi = bmp_offs[v] + threadIdx.x; wi < bmp_offs[v + 1]; wi += blockDim.x) {
                private_count += __popc(bmp_words[wi] & bitmap[bmp_word_indices[wi]]);
            }
        } else {
            for (uint32_t dst_idx = d_offsets[v] + threadIdx.x; dst_idx < d_offsets[v + 1]; dst_idx += blockDim.x) {
                uint32_t w = d_dsts[dst_idx];
                const uint32_t dst_nei_val = w / elem_bits;
                if ((bitmap_indexes[dst_nei_val >> BITMAP_SCALE_LOG]
                        >> ((w >> BITMAP_SCALE_LOG) & (elem_bits - 1))) & 0b1 == 1)
                    if ((bitmap[dst_nei_val] >> (w & (elem_bits - 1))) & 0b1 == 1)
                        private_count++;
            }
        }

        __syncwarp();
        /*warp-wise reduction*/
        WARP_REDUCE(private_count);
        if (threadIdx.x == 0) {
            auto edge_idx = eid[idx];
            auto prev = d_intersection_count_GPU[edge_idx];
            if (prev > level) {
                if (private_count < level) {
                    private_count = level;
                }
                d_intersection_count_GPU[edge_idx] = private_count;
                if (private_count == level) {
                    auto insert_idx = atomicAdd(next_cnt, 1);
                    next[insert_idx] = edge_idx;
                    inNext[edge_idx] = true;
                }
#ifndef LEGACY_SCAN
                // Update the Bucket.
                auto latest = private_count;
                if (latest > level && latest < bucket_level_end_) {
                    auto old_token = atomicCAS(in_bucket_window_ + edge_idx, InBucketFalse, InBucketTrue);
                    if (!old_token) {
                        auto insert_idx = atomicAdd(window_bucket_buf_size_, 1);
                        bucket_buf_[insert_idx] = edge_idx;
                    }
                }
#endif
            }
        }
    }
    __syncthreads();

    /*clean the bitmap*/
    if (end_src_in_bitmap - start_src_in_bitmap + 1 <= off_u_plus_one - off_u) {
        for (uint32_t idx = start_src_in_bitmap + tid; idx <= end_src_in_bitmap; idx += num_threads) {
            bitmap[idx] = 0;
        }
    } else {
        for (uint32_t idx = off_u + tid; idx < off_u_plus_one; idx += num_threads) {
            uint32_t src_nei = d_dsts[idx];
            bitmap[src_nei / elem_bits] = 0;
        }
    }
    __syncthreads();

    /*release the bitmap lock*/
    if (tid == 0)
        atomicCAS(&d_bitmap_states[sm_id * conc_blocks_per_SM + bitmap_ptr], 1, 0);
}
