#include "hip/hip_runtime.h"
#include "util/cuda/cuda_util.h"
#include "cuda_extern_functions.cuh"

__global__ void construct_bsr_row_ptr_per_thread(uint32_t *d_offsets, int32_t *d_dsts,
                                                 uint32_t num_vertices, uint32_t *bmp_offs) {
    uint32_t u = blockIdx.x * blockDim.x + threadIdx.x;
    if (u >= num_vertices) return;

    constexpr int word_in_bits = sizeof(uint32_t) * 8;
    auto prev_blk_id = -1;
    auto num_blks = 0;
    for (auto off = d_offsets[u]; off < d_offsets[u + 1]; off++) {
        auto v = d_dsts[off];
        int cur_blk_id = v / word_in_bits;
        if (cur_blk_id != prev_blk_id) {
            prev_blk_id = cur_blk_id;
            num_blks++;
        }
    }
    if ((d_offsets[u + 1] - d_offsets[u]) >= 16 && (d_offsets[u + 1] - d_offsets[u]) / num_blks > 2) {
        bmp_offs[u] = num_blks;
    } else {
        bmp_offs[u] = 0;
    }
}

__global__ void construct_bsr_content_per_thread(uint32_t *d_offsets, int32_t *d_dsts, uint32_t num_vertices,
                                                 uint32_t *bmp_offs, bmp_word_idx_type *bmp_word_indices,
                                                 bmp_word_type *bmp_words) {
    uint32_t u = blockIdx.x * blockDim.x + threadIdx.x;
    if (u >= num_vertices) return;

    auto size = bmp_offs[u + 1] - bmp_offs[u];
    if (size == 0) return;

    auto curr_size = 0;
    auto write_off = bmp_offs[u];
    auto bmp_word_idx_ptr = bmp_word_indices + write_off;
    auto bmp_words_ptr = bmp_words + write_off;
    auto prev_blk_id = -1;
    constexpr int word_in_bits = sizeof(uint32_t) * 8;
    for (auto off = d_offsets[u]; off < d_offsets[u + 1]; off++) {
        auto v = d_dsts[off];
        int cur_blk_id = v / word_in_bits;
        if (cur_blk_id != prev_blk_id) {
            prev_blk_id = cur_blk_id;
            bmp_words_ptr[curr_size] = 0;
            bmp_word_idx_ptr[curr_size++] = cur_blk_id;
        }
        bmp_words_ptr[curr_size - 1] |= static_cast<bmp_word_type>(1u) << (v % word_in_bits);
    }
}