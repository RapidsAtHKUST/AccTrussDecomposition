#include "hip/hip_runtime.h"
//
// Created by Bryan on 19/7/2019.
//

#include <omp.h>

#include <functional>
#include <iostream>
#include <unordered_map>
#include <climits>
#include <numeric>
#include <vector>

#include "util/util.h"
#include "util/graph/graph.h"
#include "util/log/log.h"
#include "util/timer.h"
#include "util/stat.h"
#include "util/containers/boolarray.h"
#include "util/cuda/primitives.cuh"
#include "util/serialization/pretty_print.h"
#include "cuda_extern_functions.cuh"

/*
 * Expensive operation: atomic update of a single address, totally synchronized.
 */
__inline__ __device__
void process_support(
        cuda_eid_t edge_idx, int level, int *EdgeSupport,
        int *next, int *next_cnt, bool *inNext,
        bool *in_bucket_window_, cuda_eid_t *bucket_buf_, uint32_t *window_bucket_buf_size_,
        int bucket_level_end_) {
    auto cur = atomicSub(&EdgeSupport[edge_idx], 1);
    if (cur == (level + 1)) {
        auto insert_idx = atomicAdd(next_cnt, 1);
        next[insert_idx] = edge_idx;
        inNext[edge_idx] = true;
    }
    if (cur <= level) {
        atomicAdd(&EdgeSupport[edge_idx], 1);
    }

#ifndef LEGACY_SCAN
    // Update the Bucket.
    auto latest = cur - 1;
    if (latest > level && latest < bucket_level_end_) {
        auto old_token = atomicCAS(in_bucket_window_ + edge_idx, false, true);
        if (!old_token) {
            auto insert_idx = atomicAdd(window_bucket_buf_size_, 1);
            bucket_buf_[insert_idx] = edge_idx;
        }
    }
#endif
}

/*
 * Expensive operation: relatively random access of inCurr and processed and afterwards EdgeSupport.
 */
__inline__ __device__
void PeelTriangle(
        int level, bool *inCurr, int *next, int *next_cnt, bool *inNext, //next_cnt is init as 0
        int *EdgeSupport, bool *processed,
        bool *in_bucket_window_, cuda_eid_t *bucket_buf_, uint32_t *window_bucket_buf_size_,
        int bucket_level_end_, cuda_eid_t e1_idx, cuda_eid_t e2_idx, cuda_eid_t e3_idx) {
    bool is_peel_e2 = !inCurr[e2_idx];
    bool is_peel_e3 = !inCurr[e3_idx];

    if (is_peel_e2 || is_peel_e3) {
        if ((!processed[e2_idx]) && (!processed[e3_idx])) {
            if (is_peel_e2 && is_peel_e3) {
                process_support(e2_idx, level, EdgeSupport, next, next_cnt, inNext,
                                in_bucket_window_, bucket_buf_, window_bucket_buf_size_, bucket_level_end_);
                process_support(e3_idx, level, EdgeSupport, next, next_cnt, inNext,
                                in_bucket_window_, bucket_buf_, window_bucket_buf_size_, bucket_level_end_);

            } else if (is_peel_e2) {
                if (e1_idx < e3_idx) {
                    process_support(e2_idx, level, EdgeSupport, next, next_cnt, inNext,
                                    in_bucket_window_, bucket_buf_, window_bucket_buf_size_,
                                    bucket_level_end_);
                }
            } else {
                if (e1_idx < e2_idx) {
                    process_support(e3_idx, level, EdgeSupport, next, next_cnt, inNext,
                                    in_bucket_window_, bucket_buf_, window_bucket_buf_size_,
                                    bucket_level_end_);
                }
            }
        }
    }
}

__global__
void sub_level_process(
        int level, int *curr, uint32_t curr_cnt, bool *inCurr,
        int *next, int *next_cnt, bool *inNext, //next_cnt is init as 0
        cuda_eid_t *offsets, vid_t *adj, cuda_eid_t *eid,
        CUDA_Edge *edge_list, int *EdgeSupport, bool *processed,
        bool *in_bucket_window_, cuda_eid_t *bucket_buf_, uint32_t *window_bucket_buf_size_,
        int bucket_level_end_) {

    auto tid = threadIdx.x;
    auto tnum = blockDim.x;
    auto bid = blockIdx.x;
    auto bnum = gridDim.x;

    __shared__ int size;
    extern __shared__ int shared[];
    int *e1_arr = shared;
    int *e2_arr = shared + tnum * 2;
    int *e3_arr = shared + tnum * 2 * 2;
    if (tid == 0) {
        size = 0;
    }
    __syncthreads();
    /*block-wise process*/
    for (auto i = bid; i < curr_cnt; i += bnum) {
        auto e1_idx = curr[i];
        CUDA_Edge e1 = edge_list[e1_idx];
        vid_t u = e1.u;
        vid_t v = e1.v;

        int u_start = offsets[u], u_end = offsets[u + 1];
        int v_start = offsets[v], v_end = offsets[v + 1];

        if (u_end - u_start > v_end - v_start) {
            swap_ele(u, v);
            swap_ele(u_start, v_start);
            swap_ele(u_end, v_end);
        }

        /*u neighbor set is smaller than v neighbor*/
        for (auto t = u_start + tid; t < u_start + (u_end - u_start + tnum - 1) / tnum * tnum; t += tnum) {
            __syncthreads();
            if (size >= tnum) {
                for (auto i = tid; i < size; i += tnum) {
                    auto e1_idx = e1_arr[i];
                    auto e2_idx = eid[e2_arr[i]];
                    auto e3_idx = eid[e3_arr[i]];

                    PeelTriangle(level, inCurr,
                                 next, next_cnt, inNext, //next_cnt is init as 0
                                 EdgeSupport, processed,
                                 in_bucket_window_, bucket_buf_, window_bucket_buf_size_, bucket_level_end_,
                                 e1_idx, e2_idx, e3_idx);
                }
                __syncthreads();
                if (tid == 0) {
                    size = 0;
                }
                __syncthreads();
            }

            int match = t >= u_end ? INT_INVALID : binary_search(adj[t], adj, v_start, v_end - 1);
            if (match != INT_INVALID) {
                auto pos = atomicAdd(&size, 1);
                e1_arr[pos] = e1_idx;
                e2_arr[pos] = t;
                e3_arr[pos] = match;
            }
        }
    }
    __syncthreads();
    for (auto i = tid; i < size; i += tnum) {
        auto e1_idx = e1_arr[i];
        auto e2_idx = eid[e2_arr[i]];
        auto e3_idx = eid[e3_arr[i]];

        PeelTriangle(level, inCurr,
                     next, next_cnt, inNext, //next_cnt is init as 0
                     EdgeSupport, processed,
                     in_bucket_window_, bucket_buf_, window_bucket_buf_size_, bucket_level_end_,
                     e1_idx, e2_idx, e3_idx);
    }
}

__global__
void update_processed(int *curr, uint32_t curr_cnt, bool *inCurr, bool *processed) {
    auto gtid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gtid < curr_cnt) {
        auto edge_off = curr[gtid];
        processed[edge_off] = true;
        inCurr[edge_off] = false;
    }
}

__global__
void output_edge_support(
        eid_t *output, int *curr, uint32_t curr_cnt,
        eid_t *edge_off_origin, eid_t start_pos) {
    auto gtid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gtid < curr_cnt) {
        output[gtid + start_pos] = edge_off_origin[curr[gtid]];
    }
}

__global__
void warp_detect_deleted_edges(
        cuda_eid_t *old_offsets, uint32_t old_offset_cnt,
        cuda_eid_t *eid, bool *old_processed,
        cuda_eid_t *histogram, bool *focus) {

    __shared__ uint32_t cnts[WARPS_PER_BLOCK];

    auto gtid = threadIdx.x + blockIdx.x * blockDim.x;
    auto gtnum = blockDim.x * gridDim.x;
    auto gwid = gtid >> WARP_BITS;
    auto gwnum = gtnum >> WARP_BITS;
    auto lane = threadIdx.x & WARP_MASK;
    auto lwid = threadIdx.x >> WARP_BITS;

    for (auto u = gwid; u < old_offset_cnt; u += gwnum) {
        if (0 == lane) cnts[lwid] = 0;
        __syncwarp();

        auto start = old_offsets[u];
        auto end = old_offsets[u + 1];
        for (auto v_idx = start + lane; v_idx < end; v_idx += WARP_SIZE) {
            auto target_edge_idx = eid[v_idx];
            focus[v_idx] = !old_processed[target_edge_idx];
            if (focus[v_idx])
                atomicAdd(&cnts[lwid], 1);
        }
        __syncwarp();

        if (0 == lane) histogram[u] = cnts[lwid];
    }
}

__global__
void filter_window(int *edge_sup, int count, bool *in_bucket, int low, int high) {
    auto gtid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gtid < count) {
        auto v = edge_sup[gtid];
        in_bucket[gtid] = (v >= low && v < high);
    }
}

__global__
void filter_with_random_append(cuda_eid_t *bucket_buf, int count, int *EdgeSupport,
                               bool *in_curr, int *curr, int *curr_cnt, int ref) {
    auto gtid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gtid < count) {
        auto edge_off = bucket_buf[gtid];
        if (EdgeSupport[edge_off] == ref) {
            in_curr[edge_off] = true;
            auto insert_idx = atomicAdd(curr_cnt, 1);
            curr[insert_idx] = edge_off;
        }
    }
}

void PKT_Scan(
        int *EdgeSupport, uint32_t edge_num, int level,
        int *curr, bool *inCurr, int &curr_cnt, cuda_eid_t *asc,
        bool *in_bucket_window_, cuda_eid_t *bucket_buf_, uint32_t *window_bucket_buf_size_, int &bucket_level_end_,
        ZLCUDAMemStat *mem_stat, ZLCUDATimer *timer) {
    static bool is_first = true;
    if (is_first) {
        hipMemset(inCurr, 0, edge_num * sizeof(bool));
#ifndef LEGACY_SCAN
        hipMemset(in_bucket_window_, 0, edge_num * sizeof(bool));
#endif
        is_first = false;
    }
#ifdef LEGACY_SCAN
    /*filter and get the bool vector*/
    long grid_size = (edge_num + BLOCK_SIZE - 1) / BLOCK_SIZE;
    execKernel(filter, grid_size, BLOCK_SIZE, timer, false, EdgeSupport, edge_num, inCurr, level);
    curr_cnt = CUBSelect(asc, curr, inCurr, edge_num, timer, mem_stat);
#else
    if (level == bucket_level_end_) {
        // Clear the bucket_removed_indicator
        bucket_level_end_ += LEVEL_SKIP_SIZE;

        long grid_size = (edge_num + BLOCK_SIZE - 1) / BLOCK_SIZE;
        execKernel(filter_window, grid_size, BLOCK_SIZE, timer, false,
                   EdgeSupport, edge_num, in_bucket_window_, level, bucket_level_end_);
        *window_bucket_buf_size_ = CUBSelect(asc, bucket_buf_, in_bucket_window_, edge_num, timer, mem_stat);
    }
    // SCAN the window.
    if (*window_bucket_buf_size_ != 0) {
        curr_cnt = 0;
        long grid_size = (*window_bucket_buf_size_ + BLOCK_SIZE - 1) / BLOCK_SIZE;
        execKernel(filter_with_random_append, grid_size, BLOCK_SIZE, timer, false,
                   bucket_buf_, *window_bucket_buf_size_, EdgeSupport, inCurr, curr, &curr_cnt, level);
    } else {
        curr_cnt = 0;
    }
    log_debug("Level: %d, curr: %'d/%'d", level, curr_cnt, *window_bucket_buf_size_);
#endif
}

double set_inter_time = 0;
double process_update_time = 0;

void PKT_LevelZeroProcess(
        int *curr, int curr_cnt, bool *inCurr,
        bool *processed,
        ZLCUDAMemStat *mem_stat,
        ZLCUDATimer *time_stat) {
    int block_size = 256;
    int grid_size = (curr_cnt + block_size - 1) / block_size;
    execKernel(
            update_processed, grid_size, block_size, time_stat, false,
            curr, curr_cnt, inCurr, processed);
}

void PKT_SubLevelProcess(
        int *EdgeSupport, eid_t *edge_off_origin,
        CUDA_Edge *edge_list, cuda_eid_t *eids, uint32_t eids_cnt, int level,
        cuda_eid_t *num_edges, vid_t *adj,
        int *curr, int curr_cnt, bool *inCurr,
        int *next, int *next_cnt, bool *inNext,
        bool *processed, bool *in_bucket_window_,
        cuda_eid_t *bucket_buf_, uint32_t *window_bucket_buf_size_, int &bucket_level_end_,
        ZLCUDAMemStat *mem_stat,
        ZLCUDATimer *time_stat) {
    int block_size = 256;
    static int shared_memory_size_per_block = block_size * sizeof(int) * 2 * 3;
    int grid_size = curr_cnt;  //each block process an edge
    Timer timer;

    /* 1st: Peeling */
    execKernelDynamicAllocation(
            sub_level_process, grid_size, block_size,
            shared_memory_size_per_block,
            time_stat, false,
            level, curr, curr_cnt, inCurr,
            next, next_cnt, inNext,
            num_edges, adj, eids,
            edge_list, EdgeSupport, processed,
            in_bucket_window_, bucket_buf_, window_bucket_buf_size_, bucket_level_end_);

    set_inter_time += timer.elapsed_and_reset();

    /* 2nd: Update the processed flags */
    grid_size = (curr_cnt + block_size - 1) / block_size;
    execKernel(
            update_processed, grid_size, block_size, time_stat, false,
            curr, curr_cnt, inCurr, processed);
    process_update_time += timer.elapsed_and_reset();
}

void PKT_SubLevelTCBased(
        cuda_graph_t &g_cuda, bool *&processed,
        int *&EdgeSupport, eid_t *&edge_off_origin, CUDA_Edge *&edge_list,
        int *&new_EdgeSupport, eid_t *&new_edge_offset_origin, CUDA_Edge *&new_edge_list,
        bool *&reversed_processed, bool *&edge_deleted, cuda_eid_t *&scanned_processed,
        cuda_eid_t *&new_offset, cuda_eid_t *&new_eid, vid_t *&new_adj,
        uint32_t edge_num, uint32_t todo,
        int *curr, int curr_cnt, bool *inCurr,
        int *next, int *next_cnt, bool *inNext,
        int level, uint32_t *&bmp_offs, bmp_word_idx_type *&bmp_word_indices, bmp_word_type *&bmp_words,
        uint32_t *d_bitmaps, uint32_t *d_bitmap_states,
        int num_words_bmp, int num_words_bmp_idx, uint32_t *vertex_count, uint32_t conc_blocks_per_SM,
        bool *&in_bucket_window_, cuda_eid_t *&bucket_buf_, cuda_eid_t *&new_bucket_buf_,
        uint32_t &window_bucket_buf_size_, int bucket_level_end_,
        ZLCUDAMemStat *mem_stat, ZLCUDATimer *time_stat) {
    auto block_size = 256;
    auto grid_size = (curr_cnt + block_size - 1) / block_size;

    /* Mark Processed */
    execKernel(update_processed, grid_size, block_size, time_stat, false, curr, curr_cnt, inCurr, processed);

    /* Shrink Edge Lists, CSR and update eid/edge_off_origin mappings */
    ShrinkCSREid(g_cuda, processed, EdgeSupport, edge_off_origin, edge_list,
                 new_EdgeSupport, new_edge_offset_origin, new_edge_list,
                 reversed_processed, edge_deleted, scanned_processed,
                 new_offset, new_eid, new_adj,
                 in_bucket_window_, bucket_buf_, new_bucket_buf_, window_bucket_buf_size_,
                 edge_num, todo, mem_stat, time_stat);

#ifndef DISABLE_BSR
    // 2nd: BSRs.
    ZLCudaMalloc(&bmp_offs, sizeof(uint32_t) * (g_cuda.n + 1), mem_stat);
    execKernel(construct_bsr_row_ptr_per_thread, (g_cuda.n + 127) / 128, 128,
               time_stat, true, g_cuda.num_edges, g_cuda.adj, g_cuda.n, bmp_offs);
    auto word_num = CUBScanExclusive(bmp_offs, bmp_offs, g_cuda.n, time_stat, mem_stat);
    bmp_offs[g_cuda.n] = word_num;
    log_info("Word Num: %d", word_num);
    ZLCudaMalloc(&bmp_word_indices, sizeof(bmp_word_idx_type) * word_num, mem_stat);
    ZLCudaMalloc(&bmp_words, sizeof(bmp_word_type) * word_num, mem_stat);
    execKernel(construct_bsr_content_per_thread, (g_cuda.n + 127) / 128, 128,
               time_stat, true, g_cuda.num_edges, g_cuda.adj, g_cuda.n, bmp_offs, bmp_word_indices, bmp_words);
#endif

    /* TC-based Support Updates */
    block_size = 1024;
    dim3 t_dimension(WARP_SIZE, block_size / WARP_SIZE); /*2-D*/
    hipMemset(vertex_count, 0, sizeof(uint32_t));

#ifndef DISABLE_BSR
    execKernelDynamicAllocation(
            bmp_bsr_update_next,
            g_cuda.n, t_dimension,
            num_words_bmp_idx * sizeof(uint32_t), time_stat, true,
            g_cuda.num_edges, g_cuda.adj, d_bitmaps, d_bitmap_states,
            vertex_count, conc_blocks_per_SM, g_cuda.eid, EdgeSupport,
            num_words_bmp, num_words_bmp_idx,
            bmp_offs, bmp_word_indices, bmp_words,
            level, next, next_cnt, inNext,
            in_bucket_window_, bucket_buf_, &window_bucket_buf_size_, bucket_level_end_);
    ZLCudaFree(bmp_offs, mem_stat);
    ZLCudaFree(bmp_word_indices, mem_stat);
    ZLCudaFree(bmp_words, mem_stat);
#else
    execKernelDynamicAllocation(
            bmp_update_next,
            g_cuda.n, t_dimension,
            num_words_bmp_idx * sizeof(uint32_t), time_stat, true,
            g_cuda.num_edges, g_cuda.adj, d_bitmaps, d_bitmap_states,
            vertex_count, conc_blocks_per_SM, g_cuda.eid, EdgeSupport,
            num_words_bmp, num_words_bmp_idx,
            level, next, next_cnt, inNext,
            in_bucket_window_, bucket_buf_, &window_bucket_buf_size_, bucket_level_end_);
#endif
}

void ShrinkCSREid(
        cuda_graph_t &g, bool *&processed,
        int *&EdgeSupport, eid_t *&edge_offset_origin, CUDA_Edge *&edge_list,
        int *&new_EdgeSupport, eid_t *&new_edge_offset_origin, CUDA_Edge *&new_edge_list,
        bool *&reversed_processed, bool *&edge_deleted, cuda_eid_t *&scanned_processed,
        cuda_eid_t *&new_offset, cuda_eid_t *&new_eid, vid_t *&new_adj,
        bool *&in_bucket_window_, cuda_eid_t *&bucket_buf_, cuda_eid_t *&new_bucket_buf_,
        uint32_t &window_bucket_buf_size_,
        uint32_t old_edge_num, uint32_t new_edge_num,
        ZLCUDAMemStat *mem_stat, ZLCUDATimer *time_stat) {
    static bool shrink_first_time = true;
    if (shrink_first_time) { //shrink first time, allocate the buffers
        shrink_first_time = false;
        Timer alloc_timer;
        ZLCudaMalloc(&new_adj, sizeof(vid_t) * new_edge_num * 2, mem_stat);
        ZLCudaMalloc(&new_eid, sizeof(eid_t) * new_edge_num * 2, mem_stat);
        ZLCudaMalloc(&new_offset, sizeof(eid_t) * (g.n + 1), mem_stat);

        ZLCudaMalloc(&edge_deleted, sizeof(bool) * old_edge_num * 2, mem_stat);
        log_info("Shrink Allocation Time: %.9lfs", alloc_timer.elapsed());
    }
    auto num_obj = mem_stat->get_num_obj();

    /*2. construct new CSR (offsets, adj) and rebuild the eid*/
    int block_size = 128;
    // Attention: new_offset gets the histogram.
    execKernel(warp_detect_deleted_edges, GRID_SIZE, block_size, time_stat, true,
               g.num_edges, g.n, g.eid, processed, new_offset, edge_deleted);

    eid_t total = CUBScanExclusive(new_offset, new_offset, g.n, time_stat, mem_stat);
    new_offset[g.n] = total;
    assert(total == new_edge_num * 2);
    hipDeviceSynchronize();
    swap_ele(g.num_edges, new_offset);

    /*new adj and eid construction*/
    CUBSelect(g.adj, new_adj, edge_deleted, old_edge_num * 2, time_stat, mem_stat);
    CUBSelect(g.eid, new_eid, edge_deleted, old_edge_num * 2, time_stat, mem_stat);
    swap_ele(g.adj, new_adj);
    swap_ele(g.eid, new_eid);

    g.m = new_edge_num * 2;
    assert(num_obj == mem_stat->get_num_obj()); //the mem object num should be the same
}

void InitBMPsBSRs(cuda_graph_t &g_cuda, uint32_t *&d_bitmaps, uint32_t *&d_bitmap_states, uint32_t *&d_vertex_count,
                  uint32_t *&bmp_offs, bmp_word_idx_type *&bmp_word_indices, bmp_word_type *&bmp_words,
                  ZLCUDAMemStat *mem_stat, ZLCUDATimer *time_stat) {
    // 1st: BMPs.
    InitBMP(&g_cuda, d_bitmaps, d_bitmap_states, d_vertex_count, mem_stat);
}

void PrepareCSRELEidQueues(graph_t *g, cuda_graph_t &g_cuda, Edge *edgeIdToEdge, CUDA_Edge *&edge_list,
                           int *&next_cnt, int *&curr, bool *&inCurr, int *&next, bool *&inNext, bool *&processed,
                           eid_t *edge_off_origin_cpu, eid_t *&edge_off_origin, cuda_eid_t *&identity_arr_asc,
                           ZLCUDAMemStat *mem_stat, ZLCUDATimer *time_stat) {
    // 1st: CSR/Eid/Edge List.
    g_cuda.n = g->n;
    g_cuda.m = g->m;
    uint32_t edge_num = g_cuda.m / 2;

    ZLCudaMalloc(&g_cuda.adj, sizeof(vid_t) * g_cuda.m, mem_stat);
    ZLCudaMalloc(&g_cuda.num_edges, sizeof(cuda_eid_t) * (g_cuda.n + 1), mem_stat);
    ZLCudaMalloc(&g_cuda.eid, sizeof(cuda_eid_t) * g_cuda.m, mem_stat);
    ZLCudaMalloc(&edge_list, sizeof(CUDA_Edge) * edge_num, mem_stat);

    hipMemcpy(g_cuda.adj, g->adj, sizeof(vid_t) * g_cuda.m, hipMemcpyHostToDevice);
//    hipMemcpy(g_cuda.num_edges, g->num_edges, sizeof(eid_t) * (g_cuda.n + 1), hipMemcpyHostToDevice);
//    hipMemcpy(g_cuda.eid, g->eid, sizeof(eid_t) * g_cuda.m, hipMemcpyHostToDevice);
#pragma omp parallel
    {
#pragma omp for
        for (auto i = 0; i < g_cuda.n + 1; i++) {
            g_cuda.num_edges[i] = g->num_edges[i];
        }
        for (cuda_eid_t i = 0; i < g_cuda.m; i++) {
            g_cuda.eid[i] = g->eid[i];
        }
    }
    hipMemcpy(edge_list, edgeIdToEdge, sizeof(CUDA_Edge) * edge_num, hipMemcpyHostToDevice);

    // 2nd: Processed.
    ZLCudaMalloc(&processed, sizeof(bool) * edge_num, mem_stat);

    // 3rd: Queue Related.
    ZLCudaMalloc(&next_cnt, sizeof(uint32_t), mem_stat);
    ZLCudaMalloc(&curr, sizeof(int) * edge_num, mem_stat);
    ZLCudaMalloc(&next, sizeof(int) * edge_num, mem_stat);
    ZLCudaMalloc(&inCurr, sizeof(bool) * edge_num, mem_stat);
    ZLCudaMalloc(&inNext, sizeof(bool) * edge_num, mem_stat);

    // 4th: Keep the edge offset mapping.
    long grid_size = (edge_num + BLOCK_SIZE - 1) / BLOCK_SIZE;
    edge_off_origin = nullptr;
    ZLCudaMalloc(&edge_off_origin, sizeof(eid_t) * edge_num, mem_stat);
    if (edge_off_origin_cpu != nullptr) {
        hipMemcpy(edge_off_origin, edge_off_origin_cpu, sizeof(eid_t) * edge_num, hipMemcpyHostToDevice);
    } else {
        execKernel(init_asc, grid_size, BLOCK_SIZE, time_stat, false, edge_off_origin, edge_num);
    }
    // 5th: Introduce identity_arr_asc for the CUB Select invocations.
    identity_arr_asc = nullptr;
    ZLCudaMalloc(&identity_arr_asc, sizeof(cuda_eid_t) * edge_num, mem_stat);
    execKernel(init_asc, grid_size, BLOCK_SIZE, time_stat, false, identity_arr_asc, edge_num);
}

void PrepareBucket(bool *&in_bucket_window_,
                   cuda_eid_t *&bucket_buf_, uint32_t *&window_bucket_buf_size_, int todo, ZLCUDAMemStat *mem_stat) {
    ZLCudaMalloc(&in_bucket_window_, sizeof(bool) * (todo + sizeof(long long)), mem_stat);

    ZLCudaMalloc(&bucket_buf_, sizeof(cuda_eid_t) * todo, mem_stat);
    ZLCudaMalloc(&window_bucket_buf_size_, sizeof(uint32_t), mem_stat);
}

void PKT_cuda(
        graph_t *g, eid_t *edge_off_origin_cpu, int *&EdgeSupport, Edge *edgeIdToEdge, int shrink_factor,
        eid_t *output, eid_t *level_start_pos, ZLCUDAMemStat *mem_stat, ZLCUDATimer *time_stat, int level) {
    extern double tc_time;
    Timer scan_timer, sub_process_timer, copy_timer, tc_timer, shrink_timer, prepare_timer;
    double scan_time = 0, sub_process_time = 0, copy_time = 0, shrink_time = 0, prepare_time = 0, penalty_tc_time = 0;
    // 1st: Prepare CSR/EL/Eid/Queues.
    cuda_graph_t g_cuda;
    CUDA_Edge *edge_list = nullptr;
    int *curr = nullptr, *next = nullptr;
    bool *inCurr = nullptr, *inNext = nullptr;
    int *curr_cnt_ptr = nullptr, *next_cnt = nullptr;
    bool *processed = nullptr;
    eid_t *edge_off_origin;
    cuda_eid_t *identity_arr_asc;

    ZLCudaMalloc(&curr_cnt_ptr, sizeof(uint32_t), mem_stat);
    auto &curr_cnt = *curr_cnt_ptr;
    curr_cnt = 0;
    PrepareCSRELEidQueues(g, g_cuda, edgeIdToEdge, edge_list,
                          next_cnt, curr, inCurr, next, inNext, processed,
                          edge_off_origin_cpu, edge_off_origin, identity_arr_asc, mem_stat, time_stat);
    uint32_t edge_num = g_cuda.m / 2;

    /* 2nd: Prepare for double buffered: CSR/EL/Eid/ES/offset-mapping/auxiliaries */
    cuda_eid_t *new_offset = nullptr;
    vid_t *new_adj = nullptr;
    CUDA_Edge *new_edge_list = nullptr;
    cuda_eid_t *new_eid = nullptr;
    int *new_EdgeSupport = nullptr;
    eid_t *new_edge_offset_origin = nullptr;
    bool *reversed_processed = nullptr;     // Auxiliaries for shrinking graphs.
    bool *edge_deleted = nullptr;           // Auxiliaries for shrinking graphs.
    cuda_eid_t *scanned_processed = nullptr;     // Auxiliaries for shrinking graphs.

    // Init Buckets.
    // Bucket Related.
    int bucket_level_end_ = level;
    bool *in_bucket_window_;
    cuda_eid_t *bucket_buf_;
    cuda_eid_t *new_bucket_buf_;
    uint32_t *window_bucket_buf_size_;
#ifndef LEGACY_SCAN
    PrepareBucket(in_bucket_window_, bucket_buf_, window_bucket_buf_size_, edge_num, mem_stat);
#endif
    // 3rd: Init Triangle-Counting-Based Support Update Data Structures (BMPs and BSRs).
    uint32_t *d_bitmaps, *d_bitmap_states, *d_vertex_count;
    uint32_t *bmp_offs = nullptr;
    bmp_word_idx_type *bmp_word_indexes = nullptr;
    bmp_word_type *bmp_words = nullptr;
    HIPContext context;
    auto conc_blocks_per_SM = context.GetConCBlocks(1024);
    InitBMPsBSRs(g_cuda, d_bitmaps, d_bitmap_states, d_vertex_count,
                 bmp_offs, bmp_word_indexes, bmp_words, mem_stat, time_stat);
    prepare_time = prepare_timer.elapsed();

    // 4th: Init Others.
    double shrink_kernel_time = 0;
    ZLCUDATimer shrink_kernel_timer;
    auto todo = edge_num;
    const auto todo_original = edge_num;
    auto deleted_acc = 0;
    auto shrink_cnt = 0;
    bool shrink_first_time = true;  //if true, the identity_arr_asc array should not be freed since it is shared with edge_off_origin
    hipDeviceSynchronize();

    time_stat->reset();
    vector <pair<int, double>> tc_stat;
    vector <pair<int, double>> shrink_stat;
    // Begin of Level-Processing, finding edges in k-truss but not in the (k+1)-truss.
    while (todo > 0) {
        log_trace("Level: %d, todo(origin): %d, todo(cur): %d., have: %d", level, todo_original, todo,
                  level_start_pos[level]);
        // 1st: Shrinking.
        if ((deleted_acc * 1.0 / todo_original) > (1.0 / shrink_factor)) { //need to shrink the graph
//        if (false) { //need to shrink the graph
            shrink_timer.reset();
            shrink_kernel_timer.reset();
            ShrinkCSREid(g_cuda, processed,
                         EdgeSupport, edge_off_origin, edge_list,
                         new_EdgeSupport, new_edge_offset_origin, new_edge_list,
                         reversed_processed, edge_deleted, scanned_processed,
                         new_offset, new_eid, new_adj,
                         in_bucket_window_, bucket_buf_, new_bucket_buf_, *window_bucket_buf_size_,
                         edge_num, todo, mem_stat, &shrink_kernel_timer);
            shrink_kernel_time += shrink_kernel_timer.elapsed();
            edge_num = todo;
            auto temp_shrink_time = shrink_timer.elapsed();

            shrink_stat.emplace_back(level, temp_shrink_time);
            shrink_time += temp_shrink_time;
            shrink_cnt++;
            deleted_acc = 0;

            shrink_first_time = false;
            log_trace("Shrink graph finished");
        }
        hipDeviceSynchronize();

        // 2nd: Frontier Generation.
        scan_timer.reset();
        PKT_Scan(EdgeSupport,
                 todo_original,
                 level, curr, inCurr, curr_cnt, identity_arr_asc,
                 in_bucket_window_, bucket_buf_, window_bucket_buf_size_, bucket_level_end_, mem_stat, time_stat);
        hipDeviceSynchronize();
        scan_time += scan_timer.elapsed();

        // 3rd: Iterative Sub-Level Processing.
        int level_acc_cnt = 0;
        while (curr_cnt > 0) {
            /* 1st: Copy this iteration results (edges to be marked as processed) to the output array */
            copy_timer.reset();
            todo -= curr_cnt;
            deleted_acc += curr_cnt;
            auto grid_size = (curr_cnt + BLOCK_SIZE - 1) / BLOCK_SIZE;
            execKernel(output_edge_support, grid_size, BLOCK_SIZE, time_stat, false, output, curr, curr_cnt,
                       edge_off_origin, level_start_pos[level] + level_acc_cnt);
            level_acc_cnt += curr_cnt;
            copy_time += copy_timer.elapsed();
            // No need to process the last level with PKT_SubLevelProcess.
            if (0 == todo) {
                break;
            }

            /* 2nd: Sub-Level Processing... */
            *next_cnt = 0;
            sub_process_timer.reset();
            hipDeviceSynchronize();
            if (level == 0) {
                PKT_LevelZeroProcess(curr, curr_cnt, inCurr, processed, mem_stat, time_stat);
            } else {
                size_t task_size = curr_cnt * (size_t)(level + 1);
                size_t left_edge_size = todo;
                double estimated_tc_time = left_edge_size / (g->m / 2.0) * tc_time + penalty_tc_time;
                double estimated_process_throughput = 4.0 * pow(10, 9);
                double estimated_peel_time = task_size / estimated_process_throughput;
                if (estimated_tc_time > estimated_peel_time) {
                    PKT_SubLevelProcess(EdgeSupport, edge_off_origin,
                                        edge_list, g_cuda.eid, edge_num, level,
                                        g_cuda.num_edges, g_cuda.adj,
                                        curr, curr_cnt, inCurr,
                                        next, next_cnt, inNext,
                                        processed,
                                        in_bucket_window_, bucket_buf_, window_bucket_buf_size_, bucket_level_end_,
                                        mem_stat, time_stat);
                } else {
                    shrink_first_time = false;
                    tc_timer.reset();
                    const uint32_t elem_bits = sizeof(uint32_t) * 8; /*#bits in a bitmap element*/
                    const uint32_t num_words_bmp = (g->n + elem_bits - 1) / elem_bits;
                    const uint32_t num_words_bmp_idx = (num_words_bmp + BITMAP_SCALE - 1) / BITMAP_SCALE;
                    PKT_SubLevelTCBased(g_cuda, processed,
                                        EdgeSupport, edge_off_origin, edge_list,
                                        new_EdgeSupport, new_edge_offset_origin, new_edge_list,
                                        reversed_processed, edge_deleted, scanned_processed,
                                        new_offset, new_eid, new_adj,
                                        edge_num, todo, curr, curr_cnt, inCurr, next, next_cnt, inNext,
                                        level, bmp_offs, bmp_word_indexes, bmp_words, d_bitmaps, d_bitmap_states,
                                        num_words_bmp, num_words_bmp_idx, d_vertex_count, conc_blocks_per_SM,
                                        in_bucket_window_, bucket_buf_, new_bucket_buf_, *window_bucket_buf_size_,
                                        bucket_level_end_,
                                        mem_stat, time_stat);
                    auto cost = tc_timer.elapsed();
                    if (estimated_tc_time * 1.2 < cost) {
                        penalty_tc_time += cost - estimated_tc_time;
                        log_info("Penalty TC-Time: %.9lfs", penalty_tc_time);
                    }
                    tc_stat.emplace_back(level, cost);
                    log_info("TC time: %.9lfs", cost);
                    edge_num = todo;
                    shrink_cnt++;
                    deleted_acc = 0;
                }
            }
            hipDeviceSynchronize();

//            log_info("curr_cnt: %d, next_cnt: %d, todo: %d.", curr_cnt, *next_cnt, todo);
            swap(curr, next);
            swap(inCurr, inNext);
            curr_cnt = *next_cnt;
            sub_process_time += sub_process_timer.elapsed();
        }

        level_start_pos[level + 1] = level_start_pos[level] + level_acc_cnt;
        level++;
    }

    hipDeviceSynchronize();
    float pkt_cuda_time = time_stat->elapsed();
    ZLCudaFree(processed, mem_stat);
    ZLCudaFree(next_cnt, mem_stat);
    ZLCudaFree(curr, mem_stat);
    ZLCudaFree(next, mem_stat);
    ZLCudaFree(inCurr, mem_stat);
    ZLCudaFree(inNext, mem_stat);

    ZLCudaFree(g_cuda.adj, mem_stat);
    ZLCudaFree(g_cuda.num_edges, mem_stat);
    ZLCudaFree(g_cuda.eid, mem_stat);
    ZLCudaFree(edge_list, mem_stat);
    ZLCudaFree(identity_arr_asc, mem_stat);
    if (!shrink_first_time) { //if false, the identity_arr_asc and edge_off_origin point to different array
        ZLCudaFree(edge_off_origin, mem_stat);

        /*free those double buffers*/
        ZLCudaFree(new_EdgeSupport, mem_stat);
        ZLCudaFree(new_edge_offset_origin, mem_stat);
        ZLCudaFree(new_edge_list, mem_stat);
        ZLCudaFree(new_offset, mem_stat);
        ZLCudaFree(new_adj, mem_stat);
        ZLCudaFree(new_eid, mem_stat);
        ZLCudaFree(reversed_processed, mem_stat);
        ZLCudaFree(edge_deleted, mem_stat);
        ZLCudaFree(scanned_processed, mem_stat);
    }
    log_info("PKT_CUDA kernel time: %.4f s.", pkt_cuda_time * 1.0 / 1000);
    log_info("Prepare CPU time: %.4f s.", prepare_time);
    log_info("Scan CPU time: %.4f s.", scan_time);

    log_info("Shrink kernel time: %.4f s.", shrink_kernel_time * 1.0 / 1000);
    log_info("Shrink CPU time: %.4f s.", shrink_time);
    log_info("Shrink cnt: %d.", shrink_cnt);
    stringstream ss;
    ss << shrink_stat;
    log_info("Shrink stat: %s.", ss.str().c_str());

    log_info("Sub process CPU time: %.4f s.", sub_process_time);
    log_info("Copy CPU time: %.4f s.", copy_time);
    log_info("Set Intersection time: %.4f s.", set_inter_time);
    stringstream ss2;
    ss2 << tc_stat;
    log_info("TC stat: %s", ss2.str().c_str());
    log_info("Update processed time: %.4f s.", process_update_time);
    log_info("Max mem size: %'lld Bytes.", mem_stat->get_max_use());
}