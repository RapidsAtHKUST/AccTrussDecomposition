#include "hip/hip_runtime.h"
#include "util/graph/graph.h"
#include "util/util.h"
#include "util/cuda/cuda_util.h"
#include "util/cuda/primitives.cuh"
#include "util/timer.h"
#include "util/primitives/primitives.h"
#include "cuda_extern_functions.cuh"

#define BSR_OPT
//#define CPU_BUILD_BSR
#ifdef CPU_BUILD_BSR
#include "cuda_tc_pack_vertex_cpu.h"
#endif

__global__ void bmp_kernel(uint32_t *d_offsets, /*card: |V|+1*/
                           int32_t *d_dsts, /*card: 2*|E|*/
                           uint32_t *d_bitmaps, /*the global bitmaps*/
                           uint32_t *d_bitmap_states, /*recording the usage of the bitmaps on the SM*/
                           uint32_t *vertex_count, /*for sequential block execution*/
                           uint32_t conc_blocks_per_SM, /*#concurrent blocks per SM*/
                           uint32_t *eid, /*card: 2*|E|*/
                           int32_t *d_intersection_count_GPU) /*card: |E|*/
{
    const uint32_t tid = threadIdx.x + blockDim.x * threadIdx.y; /*threads in a warp are with continuous threadIdx.x */
    const uint32_t tnum = blockDim.x * blockDim.y;
    const uint32_t num_nodes = gridDim.x; /*#nodes=#blocks*/
    const uint32_t elem_bits = sizeof(uint32_t) * 8; /*#bits in a bitmap element*/
    const uint32_t val_size_bitmap = (num_nodes + elem_bits - 1) / elem_bits;
    const uint32_t val_size_bitmap_indexes = (val_size_bitmap + BITMAP_SCALE - 1) >> BITMAP_SCALE_LOG;

    __shared__ uint32_t node_id, sm_id, bitmap_ptr;
    __shared__ uint32_t start_src, end_src, start_src_in_bitmap, end_src_in_bitmap;

    extern __shared__ uint32_t bitmap_indexes[];

    if (tid == 0) {
        node_id = atomicAdd(vertex_count, 1); /*get current vertex id*/
        start_src = d_offsets[node_id];
        end_src = d_offsets[node_id + 1];
        start_src_in_bitmap = d_dsts[start_src] / elem_bits;
        end_src_in_bitmap = (start_src == end_src) ? d_dsts[start_src] / elem_bits : d_dsts[end_src - 1] / elem_bits;
    } else if (tid == tnum - 1) {
        uint32_t temp = 0;
        asm("mov.u32 %0, %smid;" : "=r"(sm_id));
        /*get current SM*/
        while (atomicCAS(&d_bitmap_states[sm_id * conc_blocks_per_SM + temp], 0, 1) != 0)
            temp++;
        bitmap_ptr = temp;
    }
    /*initialize the 2-level bitmap*/
    for (uint32_t idx = tid; idx < val_size_bitmap_indexes; idx += tnum)
        bitmap_indexes[idx] = 0;
    __syncthreads();

    uint32_t *bitmap = &d_bitmaps[val_size_bitmap * (conc_blocks_per_SM * sm_id + bitmap_ptr)];

    /*construct the source node neighbor bitmap*/
    for (uint32_t idx = start_src + tid; idx < end_src; idx += tnum) {
        uint32_t src_nei = d_dsts[idx];
        const uint32_t src_nei_val = src_nei / elem_bits;
        atomicOr(&bitmap[src_nei_val], (0b1 << (src_nei & (elem_bits - 1)))); /*setting the bitmap*/
        atomicOr(&bitmap_indexes[src_nei_val >> BITMAP_SCALE_LOG],
                 (0b1 << ((src_nei >> BITMAP_SCALE_LOG) & (elem_bits - 1)))); /*setting the bitmap index*/
    }
    __syncthreads();

    /*loop the neighbors*/
    /* x dimension: warp-size
     * y dimension: number of warps
     * */
    auto du = d_offsets[node_id + 1] - d_offsets[node_id];
    for (uint32_t idx = start_src + threadIdx.y; idx < end_src; idx += blockDim.y) {
        /*each warp processes a node*/
        uint32_t private_count = 0;
        uint32_t src_nei = d_dsts[idx];
        auto dv = d_offsets[src_nei + 1] - d_offsets[src_nei];
        if (dv > du || ((du == dv) && node_id > src_nei))continue;
        uint32_t start_dst = d_offsets[src_nei];
        uint32_t end_dst = d_offsets[src_nei + 1];
        for (uint32_t dst_idx = start_dst + threadIdx.x; dst_idx < end_dst; dst_idx += blockDim.x) {
            uint32_t dst_nei = d_dsts[dst_idx];
            const uint32_t dst_nei_val = dst_nei / elem_bits;
            if ((bitmap_indexes[dst_nei_val >> BITMAP_SCALE_LOG] >> ((dst_nei >> BITMAP_SCALE_LOG) & (elem_bits - 1)))
                & 0b1 == 1)
                if ((bitmap[dst_nei_val] >> (dst_nei & (elem_bits - 1))) & 0b1 == 1)
                    private_count++;
        }
        __syncwarp();
        /*warp-wise reduction*/
        WARP_REDUCE(private_count);
        if (threadIdx.x == 0)
            d_intersection_count_GPU[eid[idx]] = private_count;
    }
    __syncthreads();

    /*clean the bitmap*/
    if (end_src_in_bitmap - start_src_in_bitmap + 1 <= end_src - start_src) {
        for (uint32_t idx = start_src_in_bitmap + tid; idx <= end_src_in_bitmap; idx += tnum) {
            bitmap[idx] = 0;
        }
    } else {
        for (uint32_t idx = start_src + tid; idx < end_src; idx += tnum) {
            uint32_t src_nei = d_dsts[idx];
            bitmap[src_nei / elem_bits] = 0;
        }
    }
    __syncthreads();

    /*release the bitmap lock*/
    if (tid == 0)
        atomicCAS(&d_bitmap_states[sm_id * conc_blocks_per_SM + bitmap_ptr], 1, 0);
}

__global__ void bmp_bsr_kernel(uint32_t *d_offsets, /*card: |V|+1*/
                               int32_t *d_dsts, /*card: 2*|E|*/
                               uint32_t *d_bitmaps, /*the global bitmaps*/
                               uint32_t *d_bitmap_states, /*recording the usage of the bitmaps on the SM*/
                               uint32_t *vertex_count, /*for sequential block execution*/
                               uint32_t conc_blocks_per_SM, /*#concurrent blocks per SM*/
                               uint32_t *eid, /*card: 2*|E|*/
                               int32_t *d_intersection_count_GPU, /*card: |E|*/
                               uint32_t *bmp_offs,
                               bmp_word_idx_type *bmp_word_indices,
                               bmp_word_type *bmp_words
) {
    const uint32_t tid = threadIdx.x + blockDim.x * threadIdx.y; /*threads in a warp are with continuous threadIdx.x */
    const uint32_t num_threads = blockDim.x * blockDim.y;
    const uint32_t num_nodes = gridDim.x; /*#nodes=#blocks*/
    constexpr uint32_t elem_bits = sizeof(uint32_t) * 8; /*#bits in a bitmap element*/
    const uint32_t val_size_bitmap = (num_nodes + elem_bits - 1) / elem_bits;
    const uint32_t val_size_bitmap_indexes = (val_size_bitmap + BITMAP_SCALE - 1) >> BITMAP_SCALE_LOG;

    __shared__ uint32_t u, sm_id, bitmap_ptr;
    __shared__ uint32_t off_u, off_u_plus_one, start_src_in_bitmap, end_src_in_bitmap;

    extern __shared__ uint32_t bitmap_indexes[];

    if (tid == 0) {
        u = atomicAdd(vertex_count, 1); /*get current vertex id*/
        off_u = d_offsets[u];
        off_u_plus_one = d_offsets[u + 1];
        start_src_in_bitmap = d_dsts[off_u] / elem_bits;
        end_src_in_bitmap = (off_u == off_u_plus_one) ? d_dsts[off_u] / elem_bits :
                            d_dsts[off_u_plus_one - 1] / elem_bits;
    } else if (tid == num_threads - 1) {
        uint32_t temp = 0;
        asm("mov.u32 %0, %smid;" : "=r"(sm_id));
        /*get current SM*/
        while (atomicCAS(&d_bitmap_states[sm_id * conc_blocks_per_SM + temp], 0, 1) != 0)
            temp++;
        bitmap_ptr = temp;
    }
    /*initialize the 2-level bitmap*/
    for (uint32_t idx = tid; idx < val_size_bitmap_indexes; idx += num_threads)
        bitmap_indexes[idx] = 0;
    __syncthreads();

    uint32_t *bitmap = &d_bitmaps[val_size_bitmap * (conc_blocks_per_SM * sm_id + bitmap_ptr)];

    /*construct the source node neighbor bitmap*/
    for (uint32_t idx = off_u + tid; idx < off_u_plus_one; idx += num_threads) {
        uint32_t v = d_dsts[idx];
        const uint32_t src_nei_val = v / elem_bits;
        atomicOr(&bitmap[src_nei_val], (0b1 << (v & (elem_bits - 1)))); /*setting the bitmap*/
        atomicOr(&bitmap_indexes[src_nei_val >> BITMAP_SCALE_LOG],
                 (0b1 << ((v >> BITMAP_SCALE_LOG) & (elem_bits - 1)))); /*setting the bitmap index*/
    }
    __syncthreads();

    auto du = d_offsets[u + 1] - d_offsets[u];
    for (uint32_t idx = off_u + threadIdx.y; idx < off_u_plus_one; idx += blockDim.y) {
        uint32_t v = d_dsts[idx];

        /*each warp processes an edge (u, v), v: v */
        auto dv = d_offsets[v + 1] - d_offsets[v];
        if (dv > du || ((du == dv) && u > v))continue;

        uint32_t private_count = 0;
        auto size_nv = bmp_offs[v + 1] - bmp_offs[v];
        if (size_nv > 0) {
            for (uint32_t wi = bmp_offs[v] + threadIdx.x; wi < bmp_offs[v + 1]; wi += blockDim.x) {
                private_count += __popc(bmp_words[wi] & bitmap[bmp_word_indices[wi]]);
            }
        } else {
            for (uint32_t dst_idx = d_offsets[v] + threadIdx.x; dst_idx < d_offsets[v + 1]; dst_idx += blockDim.x) {
                uint32_t w = d_dsts[dst_idx];
                const uint32_t dst_nei_val = w / elem_bits;
                if ((bitmap_indexes[dst_nei_val >> BITMAP_SCALE_LOG]
                        >> ((w >> BITMAP_SCALE_LOG) & (elem_bits - 1))) & 0b1 == 1)
                    if ((bitmap[dst_nei_val] >> (w & (elem_bits - 1))) & 0b1 == 1)
                        private_count++;
            }
        }

        __syncwarp();
        /*warp-wise reduction*/
        WARP_REDUCE(private_count);
        if (threadIdx.x == 0)
            d_intersection_count_GPU[eid[idx]] = private_count;
    }
    __syncthreads();

    /*clean the bitmap*/
    if (end_src_in_bitmap - start_src_in_bitmap + 1 <= off_u_plus_one - off_u) {
        for (uint32_t idx = start_src_in_bitmap + tid; idx <= end_src_in_bitmap; idx += num_threads) {
            bitmap[idx] = 0;
        }
    } else {
        for (uint32_t idx = off_u + tid; idx < off_u_plus_one; idx += num_threads) {
            uint32_t src_nei = d_dsts[idx];
            bitmap[src_nei / elem_bits] = 0;
        }
    }
    __syncthreads();

    /*release the bitmap lock*/
    if (tid == 0)
        atomicCAS(&d_bitmap_states[sm_id * conc_blocks_per_SM + bitmap_ptr], 1, 0);
}

void invoke_tc_bmp_gpu(graph_t *g, int *edge_sup) {
    ZLCUDAMemStat mem_stat;
    ZLCUDATimer time_stat;
    auto timer_ptr = &time_stat;

    // 1st: Initialization of BMP, CSR and eid.
    Timer timer;
    Timer total_timer;
    uint32_t *d_bitmaps, *d_vertex_count;
    uint32_t *d_bitmap_states;
    InitBMP(g, d_bitmaps, d_bitmap_states, d_vertex_count, &mem_stat);

    cuda_graph_t g_cuda;
    g_cuda.n = g->n;
    g_cuda.m = g->m;
    int *edge_sup_gpu;

    ZLCudaMalloc(&g_cuda.adj, sizeof(vid_t) * g_cuda.m, &mem_stat);
    ZLCudaMalloc(&g_cuda.num_edges, sizeof(eid_t) * (g_cuda.n + 1), &mem_stat);
    ZLCudaMalloc(&g_cuda.eid, sizeof(eid_t) * g_cuda.m, &mem_stat);
    ZLCudaMalloc(&edge_sup_gpu, sizeof(int) * g_cuda.m / 2, &mem_stat);

    checkCudaErrors(hipMemcpy(g_cuda.adj, g->adj, sizeof(vid_t) * g_cuda.m, hipMemcpyHostToDevice));
#pragma omp parallel
    {
#pragma omp for
        for (auto i = 0; i < g_cuda.n + 1; i++) {
            g_cuda.num_edges[i] = g->num_edges[i];
        }
#pragma omp for
        for (cuda_eid_t i = 0; i < g_cuda.m; i++) {
            g_cuda.eid[i] = g->eid[i];
        }
    }
//    checkCudaErrors(hipMemcpy(g_cuda.num_edges, g->num_edges, sizeof(eid_t) * (g_cuda.n + 1), hipMemcpyHostToDevice));
//    checkCudaErrors(hipMemcpy(g_cuda.eid, g->eid, sizeof(eid_t) * g_cuda.m, hipMemcpyHostToDevice));
    log_info("Finish BMP/CSR/eid Initialization: %.9lfs", timer.elapsed_and_reset());

    // 2nd: BSRs Initialization.
#if defined(BSR_OPT)
#if !defined(CPU_BUILD_BSR)
    uint32_t *bmp_offs;
    ZLCudaMalloc(&bmp_offs, sizeof(uint32_t) * (g->n + 1));
    execKernel(construct_bsr_row_ptr_per_thread, (g_cuda.n + 127) / 128, 128,
               timer_ptr, true, g_cuda.num_edges, g_cuda.adj, g_cuda.n, bmp_offs);
    auto word_num = CUBScanExclusive(bmp_offs, bmp_offs, g_cuda.n, &time_stat, &mem_stat);
    bmp_offs[g_cuda.n] = word_num;
    log_info("Word Num: %d", word_num);
    bmp_word_idx_type *bmp_word_indices;
    bmp_word_type *bmp_words;
    ZLCudaMalloc(&bmp_word_indices, sizeof(bmp_word_idx_type) * word_num, &mem_stat);
    ZLCudaMalloc(&bmp_words, sizeof(bmp_word_type) * word_num, &mem_stat);
    execKernel(construct_bsr_content_per_thread, (g_cuda.n + 127) / 128, 128,
               timer_ptr, true, g_cuda.num_edges, g_cuda.adj, g_cuda.n, bmp_offs, bmp_word_indices, bmp_words);
#else
    /* Construct BSRs on the CPU */
    uint32_t *bmp_offs;
    bmp_word_idx_type *bmp_word_indices;
    bmp_word_type *bmp_words;
    ConstructBSRsCPU(g, bmp_offs, bmp_word_indices, bmp_words, mem_stat, time_stat);
#endif
#endif
    log_info("Finish BSR construction: %.9lfs", timer.elapsed_and_reset());

    // 3rd: Triangle Counting.
    log_info("Launch Kernel...");
    const uint32_t elem_bits = sizeof(uint32_t) * 8; /*#bits in a bitmap element*/
    const uint32_t num_words_bmp = (g->n + elem_bits - 1) / elem_bits;
    const uint32_t num_word_bmp_idx = (num_words_bmp + BITMAP_SCALE - 1) / BITMAP_SCALE;
    uint32_t block_size = 1024; // maximally reduce the number of bitmaps
    dim3 t_dimension(WARP_SIZE, block_size / WARP_SIZE); /*2-D*/
    HIPContext context;
    auto conc_blocks_per_SM = context.GetConCBlocks(block_size);
#ifndef BSR_OPT
    execKernelDynamicAllocation(bmp_kernel, g_cuda.n, t_dimension,
                                num_word_bmp_idx * sizeof(uint32_t), timer_ptr, true,
                                g_cuda.num_edges, g_cuda.adj, d_bitmaps, d_bitmap_states,
                                d_vertex_count, conc_blocks_per_SM, g_cuda.eid, edge_sup_gpu);
#else
    execKernelDynamicAllocation(bmp_bsr_kernel, g_cuda.n, t_dimension,
                                num_word_bmp_idx * sizeof(uint32_t), timer_ptr, true,
                                g_cuda.num_edges, g_cuda.adj, d_bitmaps, d_bitmap_states,
                                d_vertex_count, conc_blocks_per_SM, g_cuda.eid, edge_sup_gpu,
                                bmp_offs, bmp_word_indices, bmp_words);
#endif
    checkCudaErrors(hipMemcpy(edge_sup, edge_sup_gpu, sizeof(int) * g_cuda.m / 2, hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());    // ensure the kernel execution finished
    log_info("Finish Support Initialization, Max mem size: %'lld Bytes, Time: %.9lfs.", mem_stat.get_max_use(),
             timer.elapsed_and_reset());

    // 4th: Free Memory.
    ZLCudaFree(d_bitmaps, &mem_stat);
    ZLCudaFree(d_bitmap_states, &mem_stat);
    ZLCudaFree(d_vertex_count, &mem_stat);

    ZLCudaFree(g_cuda.adj, &mem_stat);
    ZLCudaFree(g_cuda.num_edges, &mem_stat);
    ZLCudaFree(g_cuda.eid, &mem_stat);
    ZLCudaFree(edge_sup_gpu, &mem_stat);
    log_info("Free memory, Time: %.9lfs", timer.elapsed());
    log_info("End-To-End Time: %.9lfs", total_timer.elapsed());
}